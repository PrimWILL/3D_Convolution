
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <string.h>
using namespace std;

#define KERNEL_SIZE 5   // 크게 설정해두고 입력 kernel size만큼 제한해서 사용
#define TILE_SIZE 4     // 한 block 당 최대 thread수는 1024. TILE_SIZE가 8이상이면 BLOCK_SIZE^3>=1024이 된다.
#define BLOCK_SIZE (TILE_SIZE + (KERNEL_SIZE-1))


/* validation with results and output file */
int validation(const float *result, const float *output, int Rows, int Columns, int Depth, int kernel_size)
{
    /* check the validation */ 

   bool equal = true;

   for (int i = 0; i < Depth * Rows * Columns && equal; i++) {
      if (abs(result[i] - output[i]) >= 0.001f) {
         equal = false;
            break;
      }
   }
   if (equal) {
      printf("Results are equal!\n");
        return true;
   }
   else {
      printf("Results are NOT equal!\n");
        return false;
   }
}

/* print matrix one depth */
void print_matrix(float* matrix, int width, int height, int depth){
    printf("width*height = %d\n", width*height);
    for(int i = 0; i<width*height; i++){
        printf("%f ", matrix[i]);
        if((i+1)%width == 0) printf("\n");
        if((i+1)%(width*height) == 0) printf("\n");
    }
}



__constant__ float Mc[KERNEL_SIZE*KERNEL_SIZE*KERNEL_SIZE]; // kernel size는 입력 kernel size중 최대인 5로 잡았으며, 코드 내에서 입력 kernel size로 제한해서 사용

__global__ void convolution(float *image_d, float *kernel_d, float *output_d, int imat_x, int imat_y, int imat_z, int kernel_size){
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;

    int row_o = blockIdx.y * TILE_SIZE + ty;    
    int col_o = blockIdx.x * TILE_SIZE + tx;    
    int depth_o = blockIdx.z * TILE_SIZE + tz;

    int row_i = row_o - (kernel_size-1)/2;      // 최대 KERNEL_SIZE말고 입력받은 kernel의 size                      
    int col_i = col_o - (kernel_size-1)/2;
    int depth_i = depth_o - (kernel_size-1)/2;

    float output = 0.0f;

    __shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE][BLOCK_SIZE];    // tile -> size는 그대로

    if((depth_i>=0) && (depth_i < imat_z) && (row_i >= 0) && (row_i < imat_y) && (col_i >= 0) && (col_i < imat_x))
        Ns[tz][ty][tx] = image_d[(depth_i*imat_x*imat_y)+row_i*imat_x+col_i];
    else 
        Ns[tz][ty][tx] = 0.0f;

    __syncthreads();

    if(tz < TILE_SIZE && ty < TILE_SIZE && tx < TILE_SIZE){
        for(int k = 0; k<kernel_size; k++)
            for(int i = 0; i<kernel_size; i++)
                for(int j = 0; j<kernel_size; j++)
                    output += Mc[k*kernel_size*kernel_size+i*kernel_size+j]*Ns[k+tz][i+ty][j+tx];

        if(depth_o < imat_z && row_o < imat_y && col_o < imat_x)
            output_d[depth_o*imat_x*imat_y + row_o*imat_x+col_o] = output;
            
    }
    __syncthreads();

}

void GPU(int imat_x, int imat_y, int imat_z, float* input, int kernel_size, float* kernel, float* output){

    float *image_h, *image_d;
    float *kernel_h, *kernel_d;
    float *output_h, *output_d;

    image_h = input;
    kernel_h = kernel;
    output_h = (float*)malloc(sizeof(float)*imat_x*imat_y*imat_z);

    hipError_t err1 = hipMalloc((void**)&image_d, sizeof(float)*imat_x*imat_y*imat_z);
    hipError_t err2 = hipMalloc((void**)&kernel_d, sizeof(float)*kernel_size*kernel_size*kernel_size);
    hipError_t err3 = hipMalloc((void**)&output_d, sizeof(float)*imat_x*imat_y*imat_z);

    hipError_t err4 = hipMemcpy(image_d, image_h, sizeof(float)*imat_x*imat_y*imat_z, hipMemcpyHostToDevice);
    hipError_t err5 = hipMemcpy(kernel_d, kernel_h, sizeof(float)*kernel_size*kernel_size*kernel_size, hipMemcpyHostToDevice);
    hipError_t err6 = hipMemcpy(output_d, output_h, sizeof(float)*imat_x*imat_y*imat_z, hipMemcpyHostToDevice);

    hipError_t err7 = hipMemcpyToSymbol(HIP_SYMBOL(Mc), kernel, sizeof(float)*KERNEL_SIZE*KERNEL_SIZE*KERNEL_SIZE); 
    

    dim3 dimGrid(ceil((float)imat_x/TILE_SIZE), ceil((float)imat_y/TILE_SIZE), ceil((float)imat_z/TILE_SIZE));  // device 함수에서는 TILE_SIZE단위로 접근하기에 여기에서 BLOCK_SIZE로 접근하면 처리되지 못하는 element 발생
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);

    convolution<<<dimGrid, dimBlock>>>(image_d, kernel_d, output_d, imat_x, imat_y, imat_z, kernel_size);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
      fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        return;
    }

    hipMemcpy(output_h, output_d, sizeof(float)*imat_x*imat_y*imat_z, hipMemcpyDeviceToHost);
    //print_matrix(output_h, imat_z, imat_y, imat_x);

    /* validation with GPU and output file */
    //validation(output_h, output, imat_x, imat_y, imat_z, kernel_size);
    if (validation(output_h, output, imat_x, imat_y, imat_z, kernel_size)) printf("GPU programming valid!\n");
    else printf("GPU programming invalid! Please check one more time.\n");


    hipFree(image_d);
    hipFree(kernel_d);
    hipFree(output_d);
    free(output_h);
}


int main(int argc, char **argv)
{
    ifstream inputFile;
    ifstream kernelFile;
    ifstream outputFile;

    int imat_x, imat_y, imat_z;
    int omat_x, omat_y, omat_z;
    int kernel_size;
    float *input, *kernel, *output;
    int i = 0;

    /* get input file */
    inputFile.open(argv[1], ifstream::in);

    if (inputFile.is_open() == false) {
        cout << "The "<< argv[1] << " file can not be opend" << endl;
        return 1;
    }

    inputFile >> imat_z;
    inputFile >> imat_y;
    inputFile >> imat_x;

    input = (float*)malloc(sizeof(float) * imat_z * imat_y * imat_x);
    i = 0;
    while (!inputFile.eof()) {
        inputFile >> input[i++];
    }
    inputFile.close();


    /* get kernel file */
    kernelFile.open(argv[2], ifstream::in);

    if (kernelFile.is_open() == false) {
        cout << "The "<< argv[2] << " file can not be opend" << endl;
        return 1;
    }

    kernelFile >> kernel_size;

    kernel = (float*)malloc(sizeof(float) * kernel_size * kernel_size * kernel_size);
    i = 0;
    while (!kernelFile.eof()) {
        kernelFile >> kernel[i++];
    }
    kernelFile.close();


    /* get output file */
    outputFile.open(argv[3], ifstream::in);

    if (outputFile.is_open() == false) {
        cout << "The "<< argv[3] << " file can not be opend" << endl;
        return 1;
    }

    outputFile >> omat_z;
    outputFile >> omat_y;
    outputFile >> omat_x;

    output = (float*)malloc(sizeof(float) * omat_z * omat_y * omat_x);
    i = 0;
    while (!outputFile.eof()) {
        outputFile >> output[i++];
    }
    outputFile.close();

    /* single-thread (AVX) */

    /* multi-thread (with AVX) */

    /* GPU */ 
    GPU(imat_x, imat_y, imat_z, input, kernel_size, kernel, output);


    /* validation with single-thread and output file */
    //if (validation()) printf("single-thread valid!\n");
    //else printf("single-thread invalid! Please check one more time.\n");

    /* validation with multi-thread and output file */
    //if () printf("multi-thread valid!\n");
    //else printf("multi-thread invalid! Please check one more time.\n");

    /* validation with GPU and output file */
    //if (validation(result, output, imat_x, imat_y, imat_z, kernel_size)) printf("GPU programming valid!\n");
    //else printf("GPU programming invalid! Please check one more time.\n");

    free(input);
    free(kernel);
    free(output);

    return 0;
}